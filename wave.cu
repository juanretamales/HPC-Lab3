/*
Integrantes: Juan Retamales
*/


//#include <pmmintrin.h>

/*C library to perform Input/Output operations*/

#include <hip/hip_runtime.h>
#include <stdio.h>
/*C  library Añade funciones para convertir texto a otro formato*/
#include <stdlib.h>
#include <ctype.h>
#include <fcntl.h>

/*Libreria C para trabajar y comparar texto (de la linea de comando)*/
#include <string.h>
/* Librerias para open y write*/
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <math.h>

//incluyendo openMP
//#ifdef _OPENMP
//#include <omp.h>
//#endif

#include <time.h>

/* NOTAS
Para compilar: nvcc wave.cu -o wave.o
Para compilar2: nvcc wave.cu -o wave -lm -arch=sm_52
Para ejecutar: ./wave.o -N 256 -X 256 -Y 256 -T 100 -f salidaGrilla.raw -t 100

para probar: time ./wave.o -N 256 -X 128 -Y 128 -T 100 -f salidaGrilla.raw -t 26 > test.log


para enviar al servidor: scp code.cu jretamales@bioserver.diinf.usach.cl:/alumnos/jretamales/lab2
*/
__global__ void next(float *c_gt, float *c_gt1, float *c_gt2, int size, int t){
	int blockD = blockDim.x;
	int blockX = blockIdx.x;
	int threadX = threadIdx.x;

	//int ix = blockX * blockD + threadX;
	//if(i < values)
	//	c[i] = a[i] + b[i];
	int position = threadIdx.x + blockDim.x * blockIdx.x;
	printf("Hello Im X thread %d in block %d of %d threads and position global[%d] \n", threadX, blockX, blockD, position);
	 blockD = blockDim.y;
	 blockX = blockIdx.y;
	 threadX = threadIdx.y;
	 position = threadIdx.y + blockDim.y * blockIdx.y;
	printf("Hello Im Y thread %d in block %d of %d threads and position global[%d] \n", threadX, blockX, blockD, position);

	    float dt=0.1;
    	float dd=2.0;
    	float c=1.0;

	for(int i = 0; i<size; i++)
	{
		for(int j = 0; j<size; j++)
		{
			//para tiempo t==0
			if(t==0)
			{
				//verificacion para condicion inicial
				if((0.4*size)<i && (0.4*size)<j && i<(0.6*size) && j<(0.6*size))
				{
					c_gt[size*i+j]=20;
				}
				else
				{
					c_gt[size*i+j]=0;
				}
			}//fin if  t==0
			else
			{
				if(t==1)
				{
					if(i!=0 && j!=0 && i!=(size-1) && j!=(size-1))//verificando condion de borde
					{
						//ecuacion de Schroedinger para t=1
						c_gt[size*i+j] = c_gt1[size*i+j]+(pow(c,2)/2)*(pow((dt/dd),2))*(c_gt1[size*(i+1)+j]+c_gt1[size*(i-1)+j]+c_gt1[size*(i)+(j-1)]+c_gt1[size*(i)+(j+1)]-4*c_gt1[size*i+j]);
					}
					else
					{
						c_gt[size*i+j] = 0;
					}
				}//fin if  t==1
				else
				{// si t es mayor a 1
					if(i!=0 && j!=0 && i!=(size-1) && j!=(size-1))//verificando condion de borde
					{
						//ecuacion de Schroedinger para t>1
						c_gt[size*i+j] = 2*c_gt1[size*i+j]-c_gt2[size*i+j]+(pow(c,2))*(pow((dt/dd),2))*(c_gt1[size*(i+1)+j]+c_gt1[size*(i-1)+j]+c_gt1[size*(i)+(j-1)]+c_gt1[size*(i)+(j+1)]-4*c_gt1[size*i+j]);

					}
					else
					{
						c_gt[size*i+j] = 0;
					}
				}//fin if  t==1 else
			}//fin if  t==0 else
		}//fin for j
	}//fin for i
}

__global__ void next2(float *c_gt, float *c_gt1, float *c_gt2, int size, int t){
	int blockD = blockDim.x;
	int blockX = blockIdx.x;
	int threadX = threadIdx.x;

	//int ix = blockX * blockD + threadX;
	//if(i < values)
	//	c[i] = a[i] + b[i];
	printf("Hello Im thread %d in block %d of %d threads\n", threadX, blockX, blockD);

	    float dt=0.1;
    	float dd=2.0;
    	float c=1.0;

	for(int i = 0; i<size; i++)
	{
		for(int j = 0; j<size; j++)
		{
			//para tiempo t==0
			if(t==0)
			{
				//verificacion para condicion inicial
				if((0.4*size)<i && (0.4*size)<j && i<(0.6*size) && j<(0.6*size))
				{
					c_gt[size*i+j]=20;
				}
				else
				{
					c_gt[size*i+j]=0;
				}
			}//fin if  t==0
			else
			{
				if(t==1)
				{
					if(i!=0 && j!=0 && i!=(size-1) && j!=(size-1))//verificando condion de borde
					{
						//ecuacion de Schroedinger para t=1
						c_gt[size*i+j] = c_gt1[size*i+j]+(pow(c,2)/2)*(pow((dt/dd),2))*(c_gt1[size*(i+1)+j]+c_gt1[size*(i-1)+j]+c_gt1[size*(i)+(j-1)]+c_gt1[size*(i)+(j+1)]-4*c_gt1[size*i+j]);
					}
					else
					{
						c_gt[size*i+j] = 0;
					}
				}//fin if  t==1
				else
				{// si t es mayor a 1
					if(i!=0 && j!=0 && i!=(size-1) && j!=(size-1))//verificando condion de borde
					{
						//ecuacion de Schroedinger para t>1
						c_gt[size*i+j] = 2*c_gt1[size*i+j]-c_gt2[size*i+j]+(pow(c,2))*(pow((dt/dd),2))*(c_gt1[size*(i+1)+j]+c_gt1[size*(i-1)+j]+c_gt1[size*(i)+(j-1)]+c_gt1[size*(i)+(j+1)]-4*c_gt1[size*i+j]);

					}
					else
					{
						c_gt[size*i+j] = 0;
					}
				}//fin if  t==1 else
			}//fin if  t==0 else
		}//fin for j
	}//fin for i
}

__global__ void copyT1T(float *c_gt, float *c_gt1, int size){
	printf("\nCopianto T a T1");
	for(int i=0;i<size;i++)
	{
		for(int j=0;j<size;j++)
		{
			c_gt1[size*i+j]=c_gt[size*i+j];
		}
	}
}

__global__ void copyT2T1(float *c_gt1, float *c_gt2, int size){
	printf("\nCopianto T1 a T2");
	for(int i=0;i<size;i++)
	{
		for(int j=0;j<size;j++)
		{
			c_gt2[size*i+j]=c_gt1[size*i+j];
		}
	}
}


/*
 * Function principal encargada de recibir y gestionar los datos recibidos
 */
 int main(int argc, char *argv[])
 {
   /*Variables int guardan el archivo de salida */
   int outputF;
    /*Variables int guardan el archivo de entrada y salida respectivamente*/
    int tamanoGrilla = 0;
    int num_pasos = 0;
    int iteracionSalida = 0;
int tamanoBlockX = 0;
int tamanoBlockY = 0;

    int t, j, i;



    //creo las variables para ver el tiempo transcurrido
    clock_t start_t, end_t, total_t;
	start_t = clock();



    /*De tener menos de 5 elementos por parametros se cancela ya que es insuficiente para iniciar*/
    if (argc<4)
    {
        perror("se esperaban mas parametros...\n");
        return 0;
    }

    /*Se crea un loop para revisar los parametros recibidos por consola, como argc[0] es el nombre del ejecutable, se inicia en 1 para revisar del primer parametro*/
    for(int i=1; i<argc;i++)
    {
      if(strcmp(argv[i],"-N")==0)
      {
        /*Se verifica que el argumento posterior a -N sea un numero*/
        tamanoGrilla=atoi(argv[i+1]);
      }
	if(strcmp(argv[i],"-X")==0)
      {
        /*Se verifica que el argumento posterior a -X sea un numero*/
        tamanoBlockX=atoi(argv[i+1]);
      }
	if(strcmp(argv[i],"-Y")==0)
      {
        /*Se verifica que el argumento posterior a -Y sea un numero*/
        tamanoBlockY=atoi(argv[i+1]);
      }

      if(strcmp(argv[i],"-T")==0)
      {
        /*Se verifica que el argumento posterior a -T sea un numero*/
        num_pasos=atoi(argv[i+1]);
      }


      if(strcmp(argv[i],"-f")==0  )
      {
        /*Se verifica que el argumento posterior abriendo o creando el archivo*/
        outputF=open(argv[i+1], O_CREAT | O_WRONLY, 0600);
        if(outputF == -1)
        {
          perror("\nFailed to create an open the file.");
          //EXIT_FAILURE;
          exit(1);
        }
      }
      if(strcmp(argv[i],"-t")==0)
      {
        /*Se verifica que el argumento posterior a -t sea un numero*/
        iteracionSalida=atoi(argv[i+1]);
      }
    }/*Fin loop*/



    /*Se comprueba si llegaron todos los parametros obligatorios*/
    if(outputF != -1 && tamanoGrilla>0  && iteracionSalida>0)
    {

			dim3 numBlocks (tamanoBlockX, tamanoBlockY);//asigno el blocksize


			dim3 blocksize (tamanoGrilla / tamanoBlockX, tamanoGrilla / tamanoBlockY);


			//float grillaT2[tamanoGrilla][tamanoGrilla];//grilla en tiempo (t-2)

			//float grillaT1[tamanoGrilla][tamanoGrilla];//Grilla en tiempo (t-1)

			//float grilla[tamanoGrilla][tamanoGrilla]; //grilla en tiempo (t) actual

			float *grillaT2 = (float*)malloc(tamanoGrilla*tamanoGrilla*sizeof(float));//grilla en tiempo (t-2)
			float *grillaT1 = (float*)malloc(tamanoGrilla*tamanoGrilla*sizeof(float));//Grilla en tiempo (t-1)
			float *grilla = (float*)malloc(tamanoGrilla*tamanoGrilla*sizeof(float));//grilla en tiempo (t) actual

			float *c_gt2, *c_gt1, *c_gt;//todas las grilla cuda se guardan aqui
			hipMalloc((void**) &c_gt2, tamanoGrilla*tamanoGrilla*sizeof(float));//grilla cuda en tiempo (t-2)
			hipMalloc((void**) &c_gt1, tamanoGrilla*tamanoGrilla*sizeof(float));//Grilla cuda en tiempo (t-1)
			hipMalloc((void**) &c_gt, tamanoGrilla*tamanoGrilla*sizeof(float));//grilla cuda en tiempo (t) actual

			//copiando arreglos desde el host al device
			hipMemcpy(c_gt2, grillaT2, tamanoGrilla*tamanoGrilla*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(c_gt1, grillaT1, tamanoGrilla*tamanoGrilla*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(c_gt, grilla, tamanoGrilla*tamanoGrilla*sizeof(float), hipMemcpyHostToDevice);

			for( t=0;t<num_pasos;t++)
			{
				printf("\n   usando t=%d \n", t);
				//al final de la iteracion la grillaT1 de tiempo (t-1) pasa a ser grillaT2 que corresponde a grilla en tiempo (t-2)
				//asigno num_hebras como numero de hebras para el siguiente bloque, y asigno cuales variables son compartidas y privadas.
				next<<<numBlocks,blocksize>>>(c_gt, c_gt1, c_gt2, tamanoGrilla, t);
				hipDeviceSynchronize();
				//copiando arreglos desde el device al host
				hipMemcpy(c_gt, grilla, tamanoGrilla*sizeof(float), hipMemcpyDeviceToHost);

				//si iteracion de salida es igual al al tiempo (t) la recorro sin paralelismo e imprimo
				if(t==(iteracionSalida-1))
				{
					for( i=0;i<tamanoGrilla;i++)
					{

						for( j=0;j<tamanoGrilla;j++)
						{

							printf("\n   intentando guardar %f", grilla[tamanoGrilla*i+j]);
							write(outputF, &grilla[tamanoGrilla*i+j] , sizeof(float));
						}
					}
				}



				//al final de la iteracion la grillaT1 de tiempo (t-1) pasa a ser grillaT2 que corresponde a grilla en tiempo (t-2)
				//asigno num_hebras como numero de hebras para el siguiente bloque, y asigno cuales variables son compartidas y privadas.
        copyT2T1<<<numBlocks,blocksize>>>(c_gt1, c_gt2, tamanoGrilla);
				hipDeviceSynchronize();//sincronizo los datos
				hipMemcpy(c_gt2, c_gt1, tamanoGrilla*tamanoGrilla*sizeof(float), hipMemcpyDeviceToHost);
        //al final de la iteracion la grilla de tiempo (t) pasa a ser grillaT1 que corresponde a grilla en tiempo (t-1)
				//asigno num_hebras como numero de hebras para el siguiente bloque, y asigno cuales variables son compartidas y privadas.
        copyT1T<<<numBlocks,blocksize>>>(c_gt, c_gt1, tamanoGrilla);
				hipDeviceSynchronize();//sincronizo los datos
				//copiando arreglos desde el device al host
				hipMemcpy(c_gt, grilla, tamanoGrilla*tamanoGrilla*sizeof(float), hipMemcpyDeviceToHost);


      }//fin for t

			hipFree(c_gt2);
			hipFree(c_gt1);
			hipFree(c_gt);

      close (outputF);

	  //descomentar si se desea ver el tiempo empleado
      //printf("Tiempo usado con  Tamano[%d] num_Pasos[%d]  Salida[%d] = %f sec.\n", tamanoGrilla, num_pasos, iteracionSalida, end-start);
	end_t = clock();
	total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
   	printf("Total time taken by CPU: %f\n", (double)total_t  );
      return 0;
    }//fin if principal
  }//fin main
